#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_REPLACE_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_REPLACE(*((const double *) (const void *) (sbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))), *((double *) (void *) (dbuf + idx * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_unpack_REPLACE_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_REPLACE(*((const double *) (const void *) (sbuf + idx * sizeof(double))), *((double *) (void *) (dbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_pack_PROD_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_PROD(*((const double *) (const void *) (sbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))), *((double *) (void *) (dbuf + idx * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_unpack_PROD_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_PROD(*((const double *) (const void *) (sbuf + idx * sizeof(double))), *((double *) (void *) (dbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_pack_MAX_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_MAX_FLOAT(double, *((const double *) (const void *) (sbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))), *((double *) (void *) (dbuf + idx * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_unpack_MAX_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_MAX_FLOAT(double, *((const double *) (const void *) (sbuf + idx * sizeof(double))), *((double *) (void *) (dbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_pack_MIN_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_MIN_FLOAT(double, *((const double *) (const void *) (sbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))), *((double *) (void *) (dbuf + idx * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_unpack_MIN_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_MIN_FLOAT(double, *((const double *) (const void *) (sbuf + idx * sizeof(double))), *((double *) (void *) (dbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_pack_SUM_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_SUM(*((const double *) (const void *) (sbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))), *((double *) (void *) (dbuf + idx * sizeof(double))));
}

__global__ void yaksuri_cudai_kernel_unpack_SUM_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.child->u.hindexed.count;
    
    uintptr_t x2;
    for (intptr_t i = 0; i < md->u.contig.child->u.hindexed.count; i++) {
            uintptr_t in_elems = md->u.contig.child->u.hindexed.array_of_blocklengths[i] *
                                 md->u.contig.child->u.hindexed.child->num_elements;
            if (res < in_elems) {
                    x2 = i;
                    res %= in_elems;
                    inner_elements = md->u.contig.child->u.hindexed.child->num_elements;
                    break;
            } else {
                    res -= in_elems;
            }
    }
    
    uintptr_t x3 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    intptr_t *array_of_displs2 = md->u.contig.child->u.hindexed.array_of_displs;
    YAKSURI_CUDAI_OP_SUM(*((const double *) (const void *) (sbuf + idx * sizeof(double))), *((double *) (void *) (dbuf + x0 * extent + x1 * stride1 + array_of_displs2[x2] + x3 * sizeof(double))));
}

void yaksuri_cudai_pack_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_REPLACE_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__PROD:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_PROD_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MAX:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_MAX_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MIN:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_MIN_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__SUM:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_SUM_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

void yaksuri_cudai_unpack_contig_hindexed_double(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_REPLACE_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__PROD:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_PROD_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MAX:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_MAX_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MIN:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_MIN_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__SUM:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_SUM_contig_hindexed_double,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

