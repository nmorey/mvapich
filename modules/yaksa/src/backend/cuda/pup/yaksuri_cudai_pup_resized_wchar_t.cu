#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_REPLACE_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res;
    
    YAKSURI_CUDAI_OP_REPLACE(*((const wchar_t *) (const void *) (sbuf + x0 * extent)), *((wchar_t *) (void *) (dbuf + idx * sizeof(wchar_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_REPLACE_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res;
    
    YAKSURI_CUDAI_OP_REPLACE(*((const wchar_t *) (const void *) (sbuf + idx * sizeof(wchar_t))), *((wchar_t *) (void *) (dbuf + x0 * extent)));
}

void yaksuri_cudai_pack_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_REPLACE_resized_wchar_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

void yaksuri_cudai_unpack_resized_wchar_t(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_REPLACE_resized_wchar_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

