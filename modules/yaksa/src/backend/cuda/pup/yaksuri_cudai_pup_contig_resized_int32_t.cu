#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_BXOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_BXOR(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_BXOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_BXOR(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_REPLACE_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_REPLACE(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_REPLACE_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_REPLACE(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_LAND_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_LAND(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_LAND_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_LAND(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_PROD_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_PROD(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_PROD_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_PROD(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_BAND_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_BAND(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_BAND_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_BAND(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_LXOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_LXOR(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_LXOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_LXOR(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_MAX_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_MAX(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_MAX_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_MAX(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_BOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_BOR(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_BOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_BOR(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_LOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_LOR(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_LOR_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_LOR(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_MIN_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_MIN(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_MIN_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_MIN(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

__global__ void yaksuri_cudai_kernel_pack_SUM_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_SUM(*((const int32_t *) (const void *) (sbuf + x0 * extent + x1 * stride1)), *((int32_t *) (void *) (dbuf + idx * sizeof(int32_t))));
}

__global__ void yaksuri_cudai_kernel_unpack_SUM_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.contig.count;
    
    uintptr_t x1 = res;
    
    intptr_t stride1 = md->u.contig.child->extent;
    YAKSURI_CUDAI_OP_SUM(*((const int32_t *) (const void *) (sbuf + idx * sizeof(int32_t))), *((int32_t *) (void *) (dbuf + x0 * extent + x1 * stride1)));
}

void yaksuri_cudai_pack_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__BXOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_BXOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_REPLACE_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LAND:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_LAND_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__PROD:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_PROD_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__BAND:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_BAND_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LXOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_LXOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MAX:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_MAX_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__BOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_BOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_LOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MIN:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_MIN_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__SUM:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_SUM_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

void yaksuri_cudai_unpack_contig_resized_int32_t(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__BXOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_BXOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_REPLACE_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LAND:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_LAND_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__PROD:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_PROD_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__BAND:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_BAND_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LXOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_LXOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MAX:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_MAX_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__BOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_BOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_LOR_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__MIN:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_MIN_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__SUM:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_SUM_contig_resized_int32_t,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

