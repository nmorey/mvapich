#include "hip/hip_runtime.h"
/*
 * Copyright (C) by Argonne National Laboratory
 *     See COPYRIGHT in top-level directory
 *
 * DO NOT EDIT: AUTOMATICALLY GENERATED FILE !!
 */

#include <string.h>
#include <stdint.h>
#include <wchar.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "yaksuri_cudai_base.h"
#include "yaksuri_cudai_pup.h"

__global__ void yaksuri_cudai_kernel_pack_LXOR_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_LXOR(*((const _Bool *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + idx * sizeof(_Bool))));
}

__global__ void yaksuri_cudai_kernel_unpack_LXOR_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_LXOR(*((const _Bool *) (const void *) (sbuf + idx * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))));
}

__global__ void yaksuri_cudai_kernel_pack_LOR_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_LOR(*((const _Bool *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + idx * sizeof(_Bool))));
}

__global__ void yaksuri_cudai_kernel_unpack_LOR_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_LOR(*((const _Bool *) (const void *) (sbuf + idx * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))));
}

__global__ void yaksuri_cudai_kernel_pack_REPLACE_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_REPLACE(*((const _Bool *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + idx * sizeof(_Bool))));
}

__global__ void yaksuri_cudai_kernel_unpack_REPLACE_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_REPLACE(*((const _Bool *) (const void *) (sbuf + idx * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))));
}

__global__ void yaksuri_cudai_kernel_pack_LAND_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_LAND(*((const _Bool *) (const void *) (sbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + idx * sizeof(_Bool))));
}

__global__ void yaksuri_cudai_kernel_unpack_LAND_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, const yaksuri_cudai_md_s *__restrict__ md)
{
    const char *__restrict__ sbuf = (const char *) inbuf;
    char *__restrict__ dbuf = (char *) outbuf;
    uintptr_t extent = md->extent;
    uintptr_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uintptr_t res = idx;
    uintptr_t inner_elements = md->num_elements;
    
    if (idx >= (count * inner_elements))
        return;
    
    uintptr_t x0 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.count;
    
    uintptr_t x1 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.blocklength;
    uintptr_t x2 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.count;
    
    uintptr_t x3 = res / inner_elements;
    res %= inner_elements;
    inner_elements /= md->u.hvector.child->u.blkhindx.blocklength;
    uintptr_t x4 = res;
    
    intptr_t stride1 = md->u.hvector.stride;
    intptr_t *array_of_displs2 = md->u.hvector.child->u.blkhindx.array_of_displs;
    uintptr_t extent2 = md->u.hvector.child->extent;
    YAKSURI_CUDAI_OP_LAND(*((const _Bool *) (const void *) (sbuf + idx * sizeof(_Bool))), *((_Bool *) (void *) (dbuf + x0 * extent + x1 * stride1 + x2 * extent2 + array_of_displs2[x3] + x4 * sizeof(_Bool))));
}

void yaksuri_cudai_pack_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__LXOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_LXOR_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_LOR_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_REPLACE_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LAND:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_pack_LAND_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

void yaksuri_cudai_unpack_hvector_blkhindx__Bool(const void *inbuf, void *outbuf, uintptr_t count, yaksa_op_t op, yaksuri_cudai_md_s *md, int n_threads, int n_blocks_x, int n_blocks_y, int n_blocks_z, hipStream_t stream)
{
void *args[] = { &inbuf, &outbuf, &count, &md };
    hipError_t cerr;
    switch (op) {
        case YAKSA_OP__LXOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_LXOR_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LOR:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_LOR_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__REPLACE:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_REPLACE_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

        case YAKSA_OP__LAND:
        cerr = hipLaunchKernel((const void *) yaksuri_cudai_kernel_unpack_LAND_hvector_blkhindx__Bool,
            dim3(n_blocks_x, n_blocks_y, n_blocks_z), dim3(n_threads), args, 0, stream);
        YAKSURI_CUDAI_CUDA_ERR_CHECK(cerr);
        break;

    }
}

